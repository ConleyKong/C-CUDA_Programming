#include "hip/hip_runtime.h"
 /**
  2   this program is the first program that uses the NVIDIA GPU to do some thing
  3   alrough this thing is really easy, but it really asked GPU to do things.
  4   and this thing will be modified to a more optimized status.
  5   james leng
  6   lengjiabing
  7   2015.4.25
	test if make change can successfully updated
  8  */
  9 #include<stdio.h>
 10 #include<stdlib.h>
 11 #include<hip/hip_runtime.h>
 12 
 13 #define DATA_SIZE 1048576
 14 
 15 int data[DATA_SIZE];
 16 
 17 void GenerateNumbers(int *number, int size)
 18 {
 19     for(int i = 0; i < size; i++)
 20     {
 21         number[i] = rand() % 10;
 22     }
 23 }
 24 
 25 //this defines a global function, we it is called, this piece of code will be executed on GPU
 26 __global__ static void sumOfSquares(int *num, int * result)
 27 {
 28     int sum = 0;
 29     int i;
 30     for(i = 0; i < DATA_SIZE; i++)
 31     {
 32         sum += num[i]*num[i];
 33     }
 34 
 35     *result = sum;
 36 }

int main(){
 40 
 41 
 42     GenerateNumbers(data, DATA_SIZE);
 43 
 44 
 45     //the variables must be defined to a pointer, beacuse they are pointers assigned on CPU's memory
 46     //and they just pointed to a memory block on GPU
 47 
 48     int * gpudata, *result;
 49     hipMalloc((void**)&gpudata,sizeof(int)*DATA_SIZE);
 50     hipMalloc((void**)&result,sizeof(int));
 51     hipMemcpy(gpudata,data,sizeof(int)*DATA_SIZE,hipMemcpyHostToDevice);
 52 
 53     sumOfSquares<<<1,1,0>>>(gpudata,result);
 54 
 55     int sum;
 56     hipMemcpy(&sum,result,sizeof(int),hipMemcpyDeviceToHost);
 57     hipFree(gpudata);
 58     hipFree(result);
 59 
 60     printf("the result did by GPU is:sum:%d\n",sum);
 61 
 62     //this piece of code is used to verify the result
 63     int sumCPU = 0;
 64     for(int i = 0; i < DATA_SIZE; i++)
 65     {
 66         sumCPU += data[i]*data[i];
 67     }
 68 
 69     printf("the result did by CPU is:sum:%d\n",sumCPU);
 70 }
